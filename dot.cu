
#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <iostream>

#define WARPSIZE 32
#define TILESIZE 16
#define ELEMS_TILE 256
#define TILEDIM_BLOCK 2 //1blockあたり、16*16の小行列タイルを2*2個生成する
#define TILES_BLOCK 4
using namespace std;
using namespace nvcuda;

//128スレッド4warpで起動されることを想定。2*2のタイルを1blockで計算
//タイルできれいに分割できない行列は未対応
__global__
void dot_TensorCore(float *a, float *b, float *c, int32_t m, int32_t n, int32_t k) {

	__shared__ __half a_half[ELEMS_TILE*TILES_BLOCK] __align__(32);
	__shared__ __half b_half[ELEMS_TILE*TILES_BLOCK] __align__(32);
	__shared__ __half c_half[ELEMS_TILE*TILES_BLOCK] __align__(32);

	int32_t lid = threadIdx.x % WARPSIZE;
	int32_t lid_hex = lid % 16;
	int32_t hexid = lid / 16;
	int32_t wid = threadIdx.x / WARPSIZE;
	int32_t tileIdx_x = blockIdx.x * TILEDIM_BLOCK + wid % 2; // 自スレッドがcのx軸方向何枚目のタイル生成担当か
	int32_t tileIdx_y = blockIdx.y * TILEDIM_BLOCK + wid / 2; // 自スレッドがcのy軸以下略

	wmma::fragment<wmma::matrix_a, TILESIZE, TILESIZE, TILESIZE, __half, wmma::row_major> a_frag;
	wmma::fragment<wmma::matrix_b, TILESIZE, TILESIZE, TILESIZE, __half, wmma::row_major> b_frag;
	wmma::fragment<wmma::accumulator, TILESIZE, TILESIZE, TILESIZE, __half> c_frag;

	wmma::fill_fragment(c_frag, __float2half(0.f));
	for (int32_t i=0; i < k / TILESIZE; i++) {
		//a,bの中でのタイルの先頭要素のidx
		int32_t a_offsetbase = tileIdx_y * TILESIZE * k + i * TILESIZE;
		//16*16*16でやろうとしてるので、tidが0~15の担当要素、16~31の担当要素は隔たりがある
		//1回で小行列の2行分をa_halfに。
		a_half[wid*ELEMS_TILE + lid] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);
		a_offsetbase += 2 * k; //2行下に移動
		a_half[wid*ELEMS_TILE + lid+32] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);
		a_offsetbase += 2 * k;
		a_half[wid*ELEMS_TILE + lid+64] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);
		a_offsetbase += 2 * k;
		a_half[wid*ELEMS_TILE + lid+96] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);
		a_offsetbase += 2 * k;
		a_half[wid*ELEMS_TILE + lid+128] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);
		a_offsetbase += 2 * k;
		a_half[wid*ELEMS_TILE + lid+160] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);
		a_offsetbase += 2 * k;
		a_half[wid*ELEMS_TILE + lid+192] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);
		a_offsetbase += 2 * k;
		a_half[wid*ELEMS_TILE + lid+224] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);

		int32_t b_offsetbase = i * TILESIZE * n + tileIdx_x * TILESIZE;
		b_half[wid*ELEMS_TILE + lid] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
		b_offsetbase += 2 * n;
		b_half[wid*ELEMS_TILE + lid+32] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
		b_offsetbase += 2 * n;
		b_half[wid*ELEMS_TILE + lid+64] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
		b_offsetbase += 2 * n;
		b_half[wid*ELEMS_TILE + lid+96] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
		b_offsetbase += 2 * n;
		b_half[wid*ELEMS_TILE + lid+128] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
		b_offsetbase += 2 * n;
		b_half[wid*ELEMS_TILE + lid+160] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
		b_offsetbase += 2 * n;
		b_half[wid*ELEMS_TILE + lid+192] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
		b_offsetbase += 2 * n;
		b_half[wid*ELEMS_TILE + lid+224] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
		wmma::load_matrix_sync(a_frag, &a_half[wid*ELEMS_TILE], 16);
		wmma::load_matrix_sync(b_frag, &b_half[wid*ELEMS_TILE], 16);
		wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
	}

	wmma::store_matrix_sync(&c_half[wid*ELEMS_TILE], c_frag, 16, wmma::mem_row_major);
	int32_t c_offsetbase = tileIdx_y * TILESIZE * n + tileIdx_x * TILESIZE;
	c[c_offsetbase + hexid*n+lid_hex] = __half2float(c_half[wid*ELEMS_TILE + lid]);
	c_offsetbase += 2 * n;
	c[c_offsetbase + hexid*n+lid_hex] = __half2float(c_half[wid*ELEMS_TILE + lid+32]);
	c_offsetbase += 2 * n;
	c[c_offsetbase + hexid*n+lid_hex] = __half2float(c_half[wid*ELEMS_TILE + lid+64]);
	c_offsetbase += 2 * n;
	c[c_offsetbase + hexid*n+lid_hex] = __half2float(c_half[wid*ELEMS_TILE + lid+96]);
	c_offsetbase += 2 * n;
	c[c_offsetbase + hexid*n+lid_hex] = __half2float(c_half[wid*ELEMS_TILE + lid+128]);
	c_offsetbase += 2 * n;
	c[c_offsetbase + hexid*n+lid_hex] = __half2float(c_half[wid*ELEMS_TILE + lid+160]);
	c_offsetbase += 2 * n;
	c[c_offsetbase + hexid*n+lid_hex] = __half2float(c_half[wid*ELEMS_TILE + lid+192]);
	c_offsetbase += 2 * n;
	c[c_offsetbase + hexid*n+lid_hex] = __half2float(c_half[wid*ELEMS_TILE + lid+224]);
}

int main() {
	int32_t n = 32;
	int32_t matsize = n * n;
	float *a, *b, *c;
	float *a_dev, *b_dev, *c_dev;
	a = (float*)malloc(sizeof(float) * matsize);
	b = (float*)malloc(sizeof(float) * matsize);
	c = (float*)malloc(sizeof(float) * matsize);

	hipMalloc((void**)&a_dev, sizeof(float) * matsize);
	hipMalloc((void**)&b_dev, sizeof(float) * matsize);
	hipMalloc((void**)&c_dev, sizeof(float) * matsize);
	for (int32_t i=0; i < matsize; i++) {
		a[i] = 1.;
		b[i] = 1.;
		c[i] = 0.;
	}
	//for (int32_t i=0; i < n; i++) {
		//b[i] = 1.;
	//}
	hipMemcpy(a_dev, a, sizeof(float)*matsize, hipMemcpyHostToDevice);
	hipMemcpy(b_dev, b, sizeof(float)*matsize, hipMemcpyHostToDevice);
	dot_TensorCore<<<1, 128>>>(a_dev,  b_dev, c_dev, n, n, n);
	hipMemcpy(c, c_dev, sizeof(float)*matsize, hipMemcpyDeviceToHost);
	for (int32_t i=0; i < n; i++) {
		for (int32_t j=0; j < n; j++) {
			cout << c[i*n+j] << " ";
		}
		cout << endl;
	}
	return 0;
}
